#include <hip/hip_runtime.h>

#include <iostream>
/**
 * 总体思路：和v0一致，但是将位运算代替取余操作，取余操作耗时较大
 * 其中0和其他值按位与操作都是0
 * 提取低位 (tid & 掩码)6
 * 掩码的生成：
    掩码 (2 * index - 1) 生成一个低位为全 1、高位为 0 的二进制数。
    当 index = n 时，2 * n - 1 会生成一个二进制数，它的低 n 位是 1。例如：
    index = 1，2 * 1 - 1 = 1，二进制 0001
    index = 2，2 * 2 - 1 = 3，二进制 0011
    index = 3，2 * 3 - 1 = 7，二进制 0111
 * @tparam blockSize                                                                                                                    
 * @param d_in
 * @param d_out
 *  time latency: 0.84ms
 */
template<int blockSize>
__global__ void reduce_v0(const float *d_in, float *d_out) {
    __shared__ float shared_memory[blockSize];
    unsigned int tid = threadIdx.x;
    int global_tid = blockIdx.x * blockSize + threadIdx.x;
    // load: 每个线程加载一个元素到shared mem对应位置
    shared_memory[tid] = d_in[global_tid];
    // 涉及到shared memory的读写最好都加上__sync threads
    __syncthreads();
    for (int index = 1; index < blockDim.x; index *= 2) {
        if ((tid & (2 * index - 1)) == 0) {
            shared_memory[tid] += shared_memory[tid + index];
        }
        __syncthreads();
    }

    if (tid == 0) {
        d_out[blockIdx.x] = shared_memory[0];
    }
}

bool CheckResult(const float *out, float ground_truth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != ground_truth) {
        return false;
    }
    return true;
}

int main() {
    float millie_seconds = 0;
    const int N = 32 * 1024 * 1024;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp{};
    hipGetDeviceProperties(&deviceProp, 0);
    const int BlockSize = 256;
    const int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    auto *a = (float *) malloc(N * sizeof (float));
    float *d_a;
    hipMalloc((void **) &d_a, N * sizeof(float));

    auto *out = (float *) malloc(GridSize * sizeof (float));
    float *d_out;
    hipMalloc((void **) &d_out, GridSize * sizeof (float));

    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
    }

    int ground_truth = N * 1.0f;
    // 将初始化后的数据拷贝到GPU
    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    // 定义分配的block数量和threads数量
    dim3 Grid(GridSize);
    dim3 Block(BlockSize);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v0<BlockSize><<<Grid,Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millie_seconds, start, stop);
    // 将结果拷回CPU并check正确性
    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d", GridSize, N);
    bool is_right = CheckResult(out, ground_truth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        printf("groudtruth is: %f \n", ground_truth);
    }
    printf("reduce_v1 latency = %f ms\n", millie_seconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}