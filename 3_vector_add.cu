#include <cstdio>
#include <hip/hip_runtime.h>

typedef  float FLOAT;

/* CUDA kernel function */
__global__ void vec_add(const FLOAT *x, const FLOAT *y, FLOAT *z, int N) {
    /* 2D grid */
    unsigned int idx = (blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x);
    if (idx < N) z[idx] = y[idx] + x[idx];
}

void vec_add_cpu(const FLOAT *x, const FLOAT *y, FLOAT *z, int N)
{
    for (int i = 0; i < N; i++) z[i] = y[i] + x[i];
}

int main() {
    int N = 10000;
    unsigned int nBytes = N * sizeof(FLOAT);
    /* 1D block */
    int bs = 256;

    /* 2D grid */
    int s = ceil(sqrt((N + bs - 1.) / bs));
    dim3 grid(s, s);

    FLOAT *dx, *hx;
    FLOAT *dy, *hy;
    FLOAT *dz, *hz;

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nBytes);
    hipMalloc((void **)&dy, nBytes);
    hipMalloc((void **)&dz, nBytes);

    /* init time */
    float milliseconds = 0;

    /* allocate CPU mem */
    hx = (FLOAT *) malloc(nBytes);
    hy = (FLOAT *) malloc(nBytes);
    hz = (FLOAT *) malloc(nBytes);

    /* init */
    for (int i = 0; i < N; i++) {
        hx[i] = 1;
        hy[i] = 1;
    }

    /* copy data to GPU */
    hipMemcpy(dx, hx, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, nBytes, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    /* launch GPU kernel */
    vec_add<<<grid, bs>>>(dx, dy, dz, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    /* copy GPU result to CPU */
    hipMemcpy(hz, dz, nBytes, hipMemcpyDeviceToHost);

    /* CPU compute */
    auto* hz_cpu_res = (FLOAT *) malloc(nBytes);
    vec_add_cpu(hx, hy, hz_cpu_res, N);

    /* check GPU result with CPU*/
    for (int i = 0; i < N; ++i) {
        if (fabs(hz_cpu_res[i] - hz[i]) > 1e-6) {
            printf("Result verification failed at element index %d!\n", i);
        }
    }
    printf("Result right\n");
    hipFree(dx);
    hipFree(dy);
    hipFree(dz);

    free(hx);
    free(hy);
    free(hz);
    free(hz_cpu_res);

    return 0;
}