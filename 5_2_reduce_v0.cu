#include <hip/hip_runtime.h>

#include <iostream>
/**
 * 总体思路：
 * 通过全局id将输入数据读取到shared_memory,再通过归并思想，每个block结果被加到0号线程位置，再写回d_out
 * @tparam blockSize
 * @param d_in
 * @param d_out
 * time latency: 1.14ms
 */
template<int blockSize>
__global__ void reduce_v0(const float *d_in, float *d_out) {
    __shared__ float shared_memory[blockSize];
    unsigned int tid = threadIdx.x;
    int global_tid = blockIdx.x * blockSize + threadIdx.x;
    // load: 每个线程加载一个元素到shared mem对应位置
    shared_memory[tid] = d_in[global_tid];
    // 涉及到shared memory的读写最好都加上__sync threads
    __syncthreads();
    for (int index = 1; index < blockDim.x; index *= 2) {
        if (tid % (2 * index) == 0) {
            shared_memory[tid] += shared_memory[tid + index];
        }
        __syncthreads();
    }

    if (tid == 0) {
        d_out[blockIdx.x] = shared_memory[0];
    }
}

bool CheckResult(const float *out, float ground_truth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != ground_truth) {
        return false;
    }
    return true;
}

int main() {
    float millie_seconds = 0;
    const int N = 32 * 1024 * 1024;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp{};
    hipGetDeviceProperties(&deviceProp, 0);
    const int BlockSize = 256;
    const int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    auto *a = (float *) malloc(N * sizeof (float));
    float *d_a;
    hipMalloc((void **) &d_a, N * sizeof(float));

    auto *out = (float *) malloc(GridSize * sizeof (float));
    float *d_out;
    hipMalloc((void **) &d_out, GridSize * sizeof (float));

    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
    }

    int ground_truth = N * 1.0f;
    // 将初始化后的数据拷贝到GPU
    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    // 定义分配的block数量和threads数量
    dim3 Grid(GridSize);
    dim3 Block(BlockSize);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v0<BlockSize><<<Grid,Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millie_seconds, start, stop);
    // 将结果拷回CPU并check正确性
    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d", GridSize, N);
    bool is_right = CheckResult(out, ground_truth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        printf("groudtruth is: %f \n", ground_truth);
    }
    printf("reduce_v0 latency = %f ms\n", millie_seconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}